#include "hip/hip_runtime.h"
/********************************************************************
*  sample.cu
*  CUDA punishment with radius replacement
*********************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
//#include <cutil.h>
#include <vector>
#include <time.h>
#include <iostream>
#include <math.h>
#include <map>

using namespace std;

#ifdef _WIN32
#include <process.h>
#else
#include <unistd.h>
#endif

#define KISSRND (((((rndZ=36969*(rndZ&65535)+(rndZ>>16))<<16)+(rndW=18000*(rndW&65535)+(rndW>>16)) )^(rndY=69069*rndY+1234567))+(rndX^=(rndX<<17), rndX^=(rndX>>13), rndX^=(rndX<<5)))
#define INTABS(number) (((((0x80)<<((sizeof(int)-1)<<3))&number) ? (~number)+1 : number))

float PM[3][3]={{0.0,0.0,0.0},{0.0,0.0,0.0},{1.0,0.0,0.0}};

struct agent{
	float fitness;
	float G[2],P[3];
};

void readPMfromCL(int argc, const char * argv[]);

/************************************************************************/
/* Init CUDA                                                            */
/************************************************************************/
#if __DEVICE_EMULATION__

bool InitCUDA(int theDevice){return true;}

#else
bool InitCUDA(int theDevice)
{
	int count = 0;
	int i = 0;

	hipGetDeviceCount(&count);
	if(count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	for(i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if(prop.major >= 1) {
				break;
			}
		}
	}
	if(i == count) {
		fprintf(stderr, "There is no device supporting CUDA.\n");
		return false;
	}
//	hipSetDevice(theDevice);
//	hipSetDevice(i);
//	printf("CUDA initialized on %i.\n",i);
	return true;
}

#endif

__global__ static void makePopulationPureStrategyRandom(int popSizeX,agent* genotypeDevice,int rndW, int rndX, int rndY, int rndZ){
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	int j = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index=i*popSizeX+j;
	int t;
	rndW+=index; // make random seeds unique from one another by including index
	rndX+=index;
	rndY+=index;
	rndZ+=index;
	t=KISSRND;
	t=INTABS(t);
	switch(t%3){
		case 0:
			genotypeDevice[index].G[0]=0.0;
			genotypeDevice[index].G[1]=1.0;
		break;
		case 1:
			genotypeDevice[index].G[0]=1.0;
			genotypeDevice[index].G[1]=0.0;
		break;
		case 2:
			genotypeDevice[index].G[0]=1.0;
			genotypeDevice[index].G[1]=1.0;
		break;
	}
	float s=0.0;
	genotypeDevice[index].P[0]=genotypeDevice[index].G[0]*genotypeDevice[index].G[1];
	genotypeDevice[index].P[1]=genotypeDevice[index].G[0]*(1.0-genotypeDevice[index].G[1]);
	genotypeDevice[index].P[2]=(1.0-genotypeDevice[index].G[0])*genotypeDevice[index].G[1];
	s=genotypeDevice[index].P[0]+genotypeDevice[index].P[1]+genotypeDevice[index].P[2];
	if(s==0.0){
		for(int n=0;n<3;n++)
			genotypeDevice[index].P[n]=1.0/3.0;
	} else
		for(int n=0;n<3;n++)
			genotypeDevice[index].P[n]/=s;
}

__global__ static void makePopulationUniformlyRandom(int popSizeX,agent* genotypeDevice,int rndW, int rndX, int rndY, int rndZ){
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	int j = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index=i*popSizeX+j;
	int R[4];
	rndW+=index; // make random seeds unique from one another by including index
	rndX+=index;
	rndY+=index;
	rndZ+=index;
	for(int n=0;n<4;n++){
		int A=KISSRND;
		R[n]=INTABS(A);
	}
	rndW=R[0]; rndX=R[1]; rndY=R[2]; rndZ=R[3];
	genotypeDevice[index].fitness=0.0;
	for(int n=0;n<2;n++){
		int r=KISSRND;
		r=INTABS(r);
		genotypeDevice[index].G[n]=(float)(r&65535)/(float)65535;
	}
	float s=0.0;
	genotypeDevice[index].P[0]=genotypeDevice[index].G[0]*genotypeDevice[index].G[1];
	genotypeDevice[index].P[1]=genotypeDevice[index].G[0]*(1.0-genotypeDevice[index].G[1]);
	genotypeDevice[index].P[2]=(1.0-genotypeDevice[index].G[0])*genotypeDevice[index].G[1];
	s=genotypeDevice[index].P[0]+genotypeDevice[index].P[1]+genotypeDevice[index].P[2];
	if(s==0.0){
		for(int n=0;n<3;n++)
			genotypeDevice[index].P[n]=1.0/3.0;
	} else
		for(int n=0;n<3;n++)
			genotypeDevice[index].P[n]/=s;
}

__global__ static void computeFitness(int popSizeX,int popSize,agent* genotypeDevice,float p00,float p01,float p02,float p10,float p11,float p12,float p20,float p21,float p22){
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	int j = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index=i*popSizeX+j;
	int w,x,y;
	float fitness=0.0;
	float PM[3][3];
	PM[0][0]=p00; PM[0][1]=p01; PM[0][2]=p02;
	PM[1][0]=p10; PM[1][1]=p11; PM[1][2]=p12;
	PM[2][0]=p20; PM[2][1]=p21; PM[2][2]=p22;
	//for(w=0;w<popSize;w++)
	for(w=1;w<256;w++){
		int who=(index+w)%popSize;
		for(x=0;x<3;x++)
			for(y=0;y<3;y++)
				fitness+=PM[x][y]*(genotypeDevice[index].P[x]*genotypeDevice[who].P[y]);
	}
	genotypeDevice[index].fitness=fitness;
}

__global__ static void makeReplacement(int popSizeX,int popSize, agent *genotypeDevice,agent *replacementGenotypeDevice,float maxFit,float mutationRate,int rndW, int rndX, int rndY, int rndZ){
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	int j = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index=i*popSizeX+j;
	int ID;
	int fInt;
	float f;
	int R[4];
	rndW+=index; // make random seeds unique from one another by including index
	rndX+=index;
	rndY+=index;
	rndZ+=index;
	for(int n=0;n<4;n++){
		int A=KISSRND;
		R[n]=INTABS(A);
	}
	rndW=R[0]; rndX=R[1]; rndY=R[2]; rndZ=R[3];
	do{
		int A=KISSRND;
		ID=INTABS(A);
		ID=ID%popSize;
		fInt=KISSRND;
		fInt=(INTABS(fInt))&65535;
		f=(float)fInt/(float)65535;
	}while(f>(genotypeDevice[ID].fitness/maxFit));
	bool mutated=false;
	for(int n=0;n<2;n++){
		fInt=KISSRND;
		fInt=(INTABS(fInt))&65535;
		f=(float)fInt/(float)65535;
		if(f>=mutationRate)
			replacementGenotypeDevice[index].G[n]=genotypeDevice[ID].G[n];
		else{
			fInt=KISSRND;
			fInt=(INTABS(fInt))&65535;
			f=(float)fInt/(float)65535;
			replacementGenotypeDevice[index].G[n]=f;
		}
	}
	float s=0.0;
	replacementGenotypeDevice[index].P[0]=replacementGenotypeDevice[index].G[0]*replacementGenotypeDevice[index].G[1];
	replacementGenotypeDevice[index].P[1]=replacementGenotypeDevice[index].G[0]*(1.0-replacementGenotypeDevice[index].G[1]);
	replacementGenotypeDevice[index].P[2]=(1.0-replacementGenotypeDevice[index].G[0])*replacementGenotypeDevice[index].G[1];
	s=replacementGenotypeDevice[index].P[0]+replacementGenotypeDevice[index].P[1]+replacementGenotypeDevice[index].P[2];
	if(s==0.0){
		for(int n=0;n<3;n++)
			replacementGenotypeDevice[index].P[n]=1.0/3.0;
	} else
		for(int n=0;n<3;n++)
			replacementGenotypeDevice[index].P[n]/=s;
}
/*
__global__ static void insertReplacement(int repSizeX, int popSize, int offset, int *genotypeDevice,int *replacementGenotypeDevice){
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	int j = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index=i*repSizeX+j;
	int ID=(offset+index)%popSize;
	genotypeDevice[ID]=replacementGenotypeDevice[index];
}*/

int main(int argc, const char* argv[]){
	//very first check if we have cuda on this machine at all...
	if(!InitCUDA(0)) {
		printf("InitCUDA not working, don't know why not...\n");
		return 0;
	} else
		printf("InitCUDA passed\n");
	//let's define our variables needed
	//all memory related variables
	agent *genotypeDeviceA,*genotypeDeviceB,*swap;
	agent *genotypeHost;
	
	//other variables
	int popSizeZ=atoi(argv[2]); //30
	int popSizeX=popSizeZ*16;
	int popSizeY=popSizeZ*16;
	int popSize=popSizeX*popSizeY;
	int i,j,update;
	float maxFit;
	float Phost[3];
	float mutationRate=atof(argv[3]);//0.02
	int updates=atoi(argv[4]);
	FILE *F1=fopen(argv[1],"w+t");
	fprintf(F1,"update,p0,p1,p2\n");
	//kernel call and correct dimensions thereof
	hipError_t error;
	dim3 threadsPerBlockPop(16, 16);
	dim3 numBlocksPop(popSizeX/threadsPerBlockPop.x, popSizeY/threadsPerBlockPop.y);
	//setup cude device
	error=hipSetDevice(0);
	printf("try to set device 0\n");
	error=hipMalloc((void**)&genotypeDeviceA,sizeof(agent)*(popSize));
	if(error!=hipSuccess){
		printf("didn't work, try device 1\n");
		error=hipSetDevice(1);
		error=hipMalloc((void**)&genotypeDeviceA,sizeof(agent)*(popSize));
		if(error!=hipSuccess){
			printf("sorry could not call the right device, or not enough memory\n");
			exit(0);
		}
	}
	error=hipMalloc((void**)&genotypeDeviceB,sizeof(agent)*(popSize));
	if(error!=hipSuccess){
		printf("sorry could not allocate the copyPopulation\n");
		exit(0);
	}
	srand(time(NULL));
	//allocate the memory used on the host
	genotypeHost=(agent*)malloc(sizeof(agent)*popSize);
	if(genotypeHost==NULL){
		printf("could not allocate enough host memory...\n");
		exit(0);
	} else
		printf("alloc for host memory affirmative!\n");
	//load first genotype and phenotype into all memory
	if(mutationRate==0.0)
		makePopulationPureStrategyRandom<<<numBlocksPop,threadsPerBlockPop>>>(popSizeX,genotypeDeviceA,rand(),rand(),rand(),rand());
	else
		makePopulationUniformlyRandom<<<numBlocksPop,threadsPerBlockPop>>>(popSizeX,genotypeDeviceA,rand(),rand(),rand(),rand());
	hipDeviceSynchronize();
	readPMfromCL(argc, argv); // set the payoff matrix from command line params
	printf("population is uniform\n");
	for(update=0;update<updates;update++){
		//execute fitness computation here, NOW!!!!!!!
		computeFitness<<<numBlocksPop,threadsPerBlockPop>>>(popSizeX,popSize,genotypeDeviceA,PM[0][0],PM[0][1],PM[0][2],PM[1][0],PM[1][1],PM[1][2],PM[2][0],PM[2][1],PM[2][2]);
		hipDeviceSynchronize();
		//find max fit
		error=hipMemcpy(genotypeHost, genotypeDeviceA, sizeof(agent) * popSize, hipMemcpyDeviceToHost);
//		for(int n=0;n<20;n++)
//			printf("%i",(int)genotypeHost[n].G[0]);
//		printf("\n");
		if(error!=hipSuccess){
			printf("genotype mem copy error %s\n",hipGetErrorString(error));
			exit(0);
		}
		maxFit=0.0;
		for(j=0;j<3;j++)
			Phost[j]=0.0;
		for(i=0;i<popSize;i++){
			for(j=0;j<3;j++)
				Phost[j]+=genotypeHost[i].P[j];
			if(genotypeHost[i].fitness>maxFit)
				maxFit=genotypeHost[i].fitness;
		}
		makeReplacement<<<numBlocksPop,threadsPerBlockPop>>>(popSizeX,popSize,genotypeDeviceA,genotypeDeviceB,maxFit,mutationRate,rand(), rand(), rand(), rand());
		hipDeviceSynchronize();
		swap=genotypeDeviceA;
		genotypeDeviceA=genotypeDeviceB;
		genotypeDeviceB=swap;
		/*
		findReplacement<<<numBlocksRep,threadsPerBlockRep>>>(repSizeX,popSize,genotypeDevice,genotypeReplacementDevice,N,K,bitMask,NKTableDevice,mutationRate,rand(),rand(),rand(),rand());
		hipDeviceSynchronize();
		insertReplacement<<<numBlocksRep,threadsPerBlockRep>>>(repSizeX,popSize,rand()%popSize,genotypeDevice,genotypeReplacementDevice);
		hipDeviceSynchronize();
		*/
		float S=Phost[0]+Phost[1]+Phost[2];
		printf("update: %i maxFit:%f A:%f B:%f M:%f\n",update,maxFit,Phost[0]/S,Phost[1]/S,Phost[2]/S);
//		printf("%f %f %f - %f %f\n",genotypeHost[0].P[0],genotypeHost[0].P[1],genotypeHost[0].P[2],genotypeHost[0].G[0],genotypeHost[0].G[1]);
		fprintf(F1,"%i,%f,%f,%f\n",update,Phost[0]/S,Phost[1]/S,Phost[2]/S);
	}
	fclose(F1);
	return 0;
}

/// Reads the payoff matrix from the command line
void readPMfromCL(int argc, const char * argv[]){
	PM[0][0]=strtod(argv[5],NULL);
	PM[0][1]=strtod(argv[6],NULL);
	PM[0][2]=strtod(argv[7],NULL);
	PM[1][0]=strtod(argv[8],NULL);
	PM[1][1]=strtod(argv[9],NULL);
	PM[1][2]=strtod(argv[10],NULL);
	PM[2][0]=strtod(argv[11],NULL);
	PM[2][1]=strtod(argv[12],NULL);
	PM[2][2]=strtod(argv[13],NULL);

	FILE* fverify = fopen("verify.pm","w+t");
	//fprintf(fverify, "%i\t%i\t%i\n%i\t%i\t%i\n%i\t%i\t%i\n\n",PM[0][0],PM[0][1],PM[0][2],PM[1][0],PM[1][1],PM[1][2],PM[2][0],PM[2][1],PM[2][2]);
	fprintf(fverify, "%f\t%f\t%f\n%f\t%f\t%f\n%f\t%f\t%f",PM[0][0],PM[0][1],PM[0][2],PM[1][0],PM[1][1],PM[1][2],PM[2][0],PM[2][1],PM[2][2]);
	fclose(fverify);
}
